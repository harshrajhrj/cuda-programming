
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

#define NUM_THREADS 1000
#define NUM_BLOCKS 1000

// Kernel without atomics (incorrect)
__global__ void incrementCounterNonAtomic(int* counter) {
    // not locked
    int old = *counter;
    int new_value = old + 1;
    // not unlocked
    *counter = new_value;
}

// Kernel with atomics (correct)
__global__ void incrementCounterAtomic(int* counter) {
    int a = atomicAdd(counter, 1);
}

int main() {
    int h_counterNonAtomic = 0;
    int h_counterAtomic = 0;
    int *d_counterNonAtomic, *d_counterAtomic;

    // Allocate device memory
    hipMalloc((void**)&d_counterNonAtomic, sizeof(int));
    hipMalloc((void**)&d_counterAtomic, sizeof(int));

    // Copy initial counter values to device
    hipMemcpy(d_counterNonAtomic, &h_counterNonAtomic, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_counterAtomic, &h_counterAtomic, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernels
    incrementCounterNonAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterNonAtomic);
    incrementCounterAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterAtomic);

    // Copy results back to host
    hipMemcpy(&h_counterNonAtomic, d_counterNonAtomic, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_counterAtomic, d_counterAtomic, sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    cout<<"Non-atomic counter value: "<<h_counterNonAtomic<<"\n";
    cout<<"Atomic counter value: "<<h_counterAtomic<<"\n";

    // Free device memory
    hipFree(d_counterNonAtomic);
    hipFree(d_counterAtomic);

    return 0;
}
