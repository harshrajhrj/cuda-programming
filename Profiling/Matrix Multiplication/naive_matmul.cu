
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

#define BLOCK_SIZE 16

__global__ void matrixMultiply(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

int main() {

    // Define matrix dimensions
    const int M = 1024; // Number of rows in A and C
    const int N = 1024; // Number of columns in B and C
    const int K = 1024; // Number of columns in A and rows in B

    // Calculate matrix sizes in bytes
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    // Declare host pointers
    float *h_a, *h_b, *h_c;

    // Declare device pointers
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_a = new float[size_A];
    h_b = new float[size_B];
    h_c = new float[size_C];

    // Initialize matrices
    srand(time(NULL));
    init_matrix(h_a, M, K);
    init_matrix(h_b, K, N);

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy host to device
    hipMemcpy(d_A, h_a, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_b, size_B, hipMemcpyHostToDevice);
        
    // Kernel launch code
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
    matrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    // Synchronize device
    hipDeviceSynchronize();

    // Copy device to host
    hipMemcpy(h_c, d_C, size_C, hipMemcpyDeviceToHost);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Check for any CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    return 0;

}
