
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <nvtx3/nvToolsExt.h>
using namespace std;

#define BLOCK_SIZE 16

__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    
    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

void matrixMul(float* A, float* B, float* C, int N) {
    nvtxRangePush("Matrix Multiplication");
    
    float *d_A, *d_B, *d_C;
    int size = N * N * sizeof(float);

    nvtxRangePush("Memory Allocation");
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    nvtxRangePop();

    nvtxRangePush("Memory Copy Host2Device");
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    nvtxRangePop();

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    nvtxRangePush("Kernel Execution");
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    nvtxRangePop();

    nvtxRangePush("Memory Copy Device2Host");
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    nvtxRangePop();

    nvtxRangePush("Memory Deallocation");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    nvtxRangePop();

    nvtxRangePop();  // End of Matrix Multiplication
}

int main() {
    const int N = 1024;
    float *A = new float[N*N];
    float *B = new float[N*N];
    float *C = new float[N*N];

    // Initialize matrices A and B here...
    srand(time(NULL));
    init_matrix(A, N, N);
    init_matrix(B, N, N);

    matrixMul(A, B, C, N);

    // Use result in C...
    //for(int i = 0; i < N; i++) {
    //    for(int j = 0; j < N; j++)
    //        cout<<C[i * N + j]<<" ";
    //    cout<<endl;
    //}

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
