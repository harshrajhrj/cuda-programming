
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

const int N = 2;
// Kernel definition
__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

int main()
{
    // Kernel invocation with one block of N * N * 1 threads
    int numBlocks = 1;
    float A[N][N] = {
        {2, 2},
        {2, 2}
    };
    float B[N][N] = {
        {2, 2},
        {2, 2}
    };
    float C[N][N];
    dim3 threadsPerBlock(N, N);
    MatAdd<<<numBlocks, threadsPerBlock>>>(A, B, C);

    cout<<"";

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++)
            cout<<C[i][j]<<" ";
        cout<<endl;
    }
}